#include "hip/hip_runtime.h"
#include <tira/cuda/error.h>
#include "raytrace.h"

// CUDA device function for light calculation
__device__ glm::vec3 calculateIllumination(const light* lights, size_t lightCount, const sphere* spheres, size_t sphereCount, const hit& intersection) {
    glm::vec3 accumulatedLight(0.0f);

    for (size_t lightIndex = 0; lightIndex < lightCount; ++lightIndex) {
        ray lightRay;
        lightRay.origin = intersection.pos;
        glm::vec3 toLight = lights[lightIndex].position - intersection.pos;
        float lightDistance = glm::length(toLight);
        lightRay.direction = glm::normalize(toLight);

        // Check for occlusion
        bool isOccluded = false;
        for (size_t sphereIndex = 0; sphereIndex < sphereCount; ++sphereIndex) {
            if (&spheres[sphereIndex] == static_cast<const sphere*>(intersection.obj)) continue;
            if (spheres[sphereIndex].intersect(lightRay, lightDistance)) {
                isOccluded = true;
                break;
            }
        }

        // Accumulate light if not occluded
        if (!isOccluded) {
            float intensity = glm::dot(intersection.norm, lightRay.direction);
            if (intensity > 0) {
                accumulatedLight += intensity * lights[lightIndex].color;
            }
        }
    }

    return intersection.color * glm::clamp(accumulatedLight, 0.0f, 1.0f);
}

// CUDA kernel for tracing pixels
__global__ void traceKernel(unsigned char* outputImage, int imageWidth, int imageHeight, tira::camera camera, const sphere* spheres, size_t sphereCount, const light* lights, size_t lightCount) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= imageWidth || y >= imageHeight) return;

    float normalizedX = static_cast<float>(x) / imageWidth - 0.5f;
    float normalizedY = -static_cast<float>(y) / imageHeight + 0.5f;

    ray pixelRay;
    pixelRay.origin = camera.position();
    pixelRay.direction = camera.ray(normalizedX, normalizedY);

    hit closestIntersection;
    closestIntersection.t = 1e9f;
    bool hasHit = false;

    for (size_t sphereIndex = 0; sphereIndex < sphereCount; ++sphereIndex) {
        hit currentHit;
        if (spheres[sphereIndex].intersect(pixelRay, currentHit)) {
            hasHit = true;
            if (currentHit.t < closestIntersection.t) {
                closestIntersection = currentHit;
            }
        }
    }

    unsigned int pixelIndex = (y * imageWidth + x) * 3;
    if (hasHit) {
        glm::vec3 color = calculateIllumination(lights, lightCount, spheres, sphereCount, closestIntersection);
        outputImage[pixelIndex + 0] = static_cast<unsigned char>(color.r * 255.0f);
        outputImage[pixelIndex + 1] = static_cast<unsigned char>(color.g * 255.0f);
        outputImage[pixelIndex + 2] = static_cast<unsigned char>(color.b * 255.0f);
    } else {
        outputImage[pixelIndex + 0] = 0;
        outputImage[pixelIndex + 1] = 0;
        outputImage[pixelIndex + 2] = 0;
    }
}

// Function to manage GPU ray tracing
void gpuRayTracing(unsigned char* outputImage, int imageWidth, int imageHeight, const tira::camera& camera, const std::vector<sphere>& spheres, const std::vector<light>& lights, int cudaDeviceId) {
    size_t sphereCount = spheres.size();
    size_t lightCount = lights.size();

    hipDeviceProp_t deviceProperties;
    HANDLE_ERROR(hipGetDeviceProperties(&deviceProperties, cudaDeviceId));

    // Allocate GPU memory for image, spheres, and lights
    unsigned char* gpuImage;
    HANDLE_ERROR(hipMalloc(&gpuImage, imageWidth * imageHeight * 3));

    sphere* gpuSpheres;
    HANDLE_ERROR(hipMalloc(&gpuSpheres, sphereCount * sizeof(sphere)));
    HANDLE_ERROR(hipMemcpy(gpuSpheres, spheres.data(), sphereCount * sizeof(sphere), hipMemcpyHostToDevice));

    light* gpuLights;
    HANDLE_ERROR(hipMalloc(&gpuLights, lightCount * sizeof(light)));
    HANDLE_ERROR(hipMemcpy(gpuLights, lights.data(), lightCount * sizeof(light), hipMemcpyHostToDevice));

    // Configure CUDA kernel launch
    int threadsPerBlock = deviceProperties.maxThreadsPerBlock;
    int blockSize = static_cast<int>(sqrt(threadsPerBlock));
    dim3 blockDimensions(blockSize, blockSize);
    dim3 gridDimensions((imageWidth + blockDimensions.x - 1) / blockDimensions.x, (imageHeight + blockDimensions.y - 1) / blockDimensions.y);

    // Launch kernel
    traceKernel<<<gridDimensions, blockDimensions>>>(gpuImage, imageWidth, imageHeight, camera, gpuSpheres, sphereCount, gpuLights, lightCount);
    HANDLE_ERROR(hipDeviceSynchronize());

    // Copy results back to CPU
    HANDLE_ERROR(hipMemcpy(outputImage, gpuImage, imageWidth * imageHeight * 3, hipMemcpyDeviceToHost));

    // Free GPU memory
    HANDLE_ERROR(hipFree(gpuSpheres));
    HANDLE_ERROR(hipFree(gpuLights));
    HANDLE_ERROR(hipFree(gpuImage));
}